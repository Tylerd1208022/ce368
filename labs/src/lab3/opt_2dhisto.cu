#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <cutil.h>
#include "util.h"
#include "ref_2dhisto.h"

__global__ void histogramKernel(uint* d_result, uint* d_data, int dataN, int BIN_COUNT){ // should use BIN_COUNT somehow
    const int globalTid = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    const int BC = 1024;

    __shared__ uint s_Hist[BC];

    // initialize hist to 0
    for(int pos = threadIdx.x; pos < BC; pos += blockDim.x){
        s_Hist[pos] = 0;
    }

    __syncthreads();

    // compute hist
    for(int pos = threadIdx.x; pos < INPUT_HEIGHT * INPUT_WIDTH; pos += numThreads){
        int x = pos % INPUT_WIDTH;
        int y = pos / INPUT_WIDTH;
        uint data = d_data[x + (y * BC * 4)];
        if (s_Hist[data] < 255) atomicAdd(s_Hist + data, 1);
    }

    __syncthreads();

    // merge bins
    for(int pos = threadIdx.x * 4; pos + 4 <= BC; pos += blockDim.x * 4){
        uint merged_bin = 0;
        merged_bin += s_Hist[pos];
        merged_bin += s_Hist[pos + 1] << 8;
        merged_bin += s_Hist[pos + 2] << 16;
        merged_bin += s_Hist[pos + 3] << 24;
        atomicAdd(d_result + pos/4, merged_bin);
    }
}

void opt_2dhisto(uint* d_result, uint* d_data, int dataN, int BIN_COUNT)
{
    /* This function should only contain a call to the GPU 
       histogramming kernel. Any memory allocations and
       transfers must be done outside this function */
       dim3 blockSize = (32, 1, 1);
       dim3 gridSize = (1, 1, 1);
       histogramKernel<<<gridSize,blockSize>>>(d_result, d_data, dataN, BIN_COUNT);
}

/* Include below the implementation of any other functions you need */
uint32_t* allocateInputOnDevice(uint32_t** hostInput,int height, int width){
    //1D Alloc/*
    uint32_t* pointer = *hostInput;
    hipMalloc((void**)&pointer,4*4096*4096);
    hipMemcpy(pointer,*hostInput,sizeof(uint32_t)*4096*4096,hipMemcpyHostToDevice);
    return pointer;
    //uint32_t** rowPointers; //Empty 2DArray
    
    /*hipMalloc((void**)&rowPointers,sizeof(uint32_t*)*INPUT_HEIGHT); //Create 2d Array on GPU

    uint32_t** pointerArray = (uint32_t**) calloc(INPUT_HEIGHT, sizeof(void*)); // Create host array for row pointer
    
    for (int i = 0 ; i < height;i++){
        hipMalloc((void**)&pointerArray[i], sizeof(uint32_t) * 4096);//Create actual row vector
        //hipMemcpy(pointerArray[i], hostInput[i], sizeof(uint32_t) * INPUT_WIDTH, hipMemcpyHostToDevice);
        std::cout<<"good"<<i<<std::endl;
        std::cout<<hostInput[i][0]<<std::endl;
    }
    hipMemcpy(rowPointers, pointerArray, sizeof(uint32_t*)*INPUT_HEIGHT, hipMemcpyHostToDevice);

    free(pointerArray);
*/
    //return rowPointers;
}

uint8_t* allocateHistogramOnDevice(uint8_t** hostHisto, int height, int width){
    uint8_t* deviceHisto = *hostHisto;
    hipMalloc((void**)&deviceHisto,height*width*sizeof(uint8_t));
    hipMemcpy(deviceHisto,hostHisto,height*width*sizeof(uint32_t),hipMemcpyHostToDevice);
    return deviceHisto;
}


void cudaTeardown(uint8_t* deviceHisto, uint8_t*hostHisto, uint32_t* deviceInput){
    hipDeviceSynchronize();
    hipMemcpy(hostHisto,deviceHisto,1024*sizeof(uint8_t),hipMemcpyDeviceToHost);
    hipFree(deviceHisto);
    
    //for(int i = 0; i<INPUT_HEIGHT; i++){
    //    hipFree(&deviceInput[i]);
    //}
    hipFree(deviceInput);
}

